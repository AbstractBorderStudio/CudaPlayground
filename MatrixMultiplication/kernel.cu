﻿/*
	Daniel Bologna 2024
*/

// system includes
#include <stdio.h>

// cuda includes
#include "hip/hip_runtime.h"


int main() {
	hipError_t cudaStatus;

	// create matrices
	

	// exit app
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error Resetting Cuda Device!\n");
		return 1;
	}
	else {
		fprintf(stdout, "Successfully Reset Device\n");
	}

	return 0;
}