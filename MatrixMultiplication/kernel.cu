﻿/*
	Daniel Bologna 2024
*/

// system includes
#include <stdio.h>

// cuda includes
#include "hip/hip_runtime.h"


// main func
hipError_t MultMatrices(const int* A, const int* B, int* C, int m, int n, int k);

// kernel
__global__ void MultMatrixGPU(int* res, const int* A, const int* B, int m, int n, int k) {
	// each thread makes a vector-vector multiplication
}

int main() {
	hipError_t cudaStatus;

	// create matrices // to fix indexing!
	const int m = 2;
	const int n = 2;
	const int k = 3;
	const int A[m][k] = {
		{1,2,3},
		{1,2,3}
	};
	const int B[k][n] = {
		{1,2},
		{1,2},
		{1,2}
	};
	int C[m][m] = { 0 }; // matrices filled with zeros

	// pass pointer to the first element
	cudaStatus = MultMatrices(A[0], B[0], C[0], m, n, k);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error MultMatrices!\n");
		return 1;
	}
	else {
		fprintf(stdout, "Successfully Multiplied Matrices\n");
	}

	// print result

	// exit app
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error Resetting Cuda Device!\n");
		return 1;
	}
	else {
		fprintf(stdout, "Successfully Reset Device\n");
	}

	return 0;
}

hipError_t MultMatrices(const int* A, const int* B, int* C, int m, int n, int k)
{
	hipError_t cudaStatus = hipSuccess;

	// create cuda buffers
	const int* cuda_a;
	const int* cuda_b;
	int* cuda_c;

	int size_a = m * k;
	int size_b = k * n;
	int size_c = m * n;

	// malloc in gpu
	cudaStatus = hipMalloc((void**)&cuda_a, size_a * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error hipMalloc cuda_a!\n");
		goto Error;
	}
	else {
		fprintf(stdout, "Successfully allocated cuda_a\n");
	}

	cudaStatus = hipMalloc((void**)&cuda_b, size_b * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error hipMalloc cuda_b!\n");
		goto Error;
	}
	else {
		fprintf(stdout, "Successfully allocated cuda_b\n");
	}
	
	cudaStatus = hipMalloc((void**)&cuda_c, size_c * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error hipMalloc cuda_c!\n");
		goto Error;
	}
	else {
		fprintf(stdout, "Successfully allocated cuda_c\n");
	}

	// copy mem from host to device
	cudaStatus = hipMemcpy((void*)cuda_a, A, size_a * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error hipMemcpy cuda_a!\n");
		goto Error;
	}
	else {
		fprintf(stdout, "Successfully copied data inside cuda_a\n");
	}

	// copy mem from host to device
	cudaStatus = hipMemcpy((void*)cuda_b, B, size_b * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error hipMemcpy cuda_b!\n");
		goto Error;
	}
	else {
		fprintf(stdout, "Successfully copied data inside cuda_b\n");
	}

	// launch kernel

	// coby data back
	// copy mem from host to device
	//cudaStatus = cudaMemcpy(C, (void*)cuda_c, size_c * sizeof(int), cudaMemcpyDeviceToHost);
	//if (cudaStatus != cudaSuccess) {
	//	fprintf(stderr, "Error cudaMemcpy cuda_c!\n");
	//	goto Error;
	//}
	//else {
	//	fprintf(stdout, "Successfully copied back data inside cuda_c\n");
	//}

Error:
	hipFree(&cuda_a);
	hipFree(&cuda_b);
	hipFree(&cuda_c);

	fprintf(stdout, "Successfully cleared cuda buffers\n");

	return cudaStatus;
}
 